#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 5
#define M 6
__global__ void dkernal(unsigned int* matrix){
    /*if(threadIdx.x == 0 &&blockIdx.x == 0 &&
       threadIdx.y == 0 && blockIdx.y == 0 &&
       threadIdx.z == 0 && blockIdx.z == 0){
           printf("%d %d %d %d %d %d\n", gridDim.x, gridDim.y, gridDim.z, 
                                         blockDim.x, blockDim.y, blockDim.z);
       }*/
       /*int i = threadIdx.x; 
               matrix[threadIdx.y + (i * M)] = 
                                   threadIdx.y + (i * M) ;*/

        int j = threadIdx.x;
        int i = blockIdx.x;
        matrix[i * M + j] = i * M + j ; 
       
}

int main(){
    unsigned *matrix, *hmatrix;
    hipMalloc(&matrix, N * M * sizeof(unsigned));
    hmatrix = (unsigned*)malloc(N * M * sizeof(unsigned));
    dkernal <<<N, M>>>(matrix);
    hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), 
                                    hipMemcpyDeviceToHost);
    for(unsigned i = 0 ; i < N ; ++i)
    {
        for(unsigned j = 0 ; j < M ; ++j)
        {
            printf("%3d", hmatrix[i * M + j]);
        }
        printf("\n");
    }
    return 0;
}
