#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCKSIZE 1024
__global__ void dkernal(unsigned *vector, unsigned vectorsize)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < vectorsize) vector[id] = id;
}  

int main(int nn, char *str[])
{
    unsigned N = 1025; //atoi(str[1]);
    unsigned *vector, *hvector;
    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned*)malloc(N * sizeof(unsigned));
    
    unsigned nblocks = ceil((float)N/BLOCKSIZE);
    printf("nblocks = %d\n", nblocks);
 
    dkernal<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    for(unsigned i = 0 ; i < N ; ++i)
    {
        printf("%6d", hvector[i]);
    }
    return 0;
}
