#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 8000  

__global__ void dkernal(){
    if(threadIdx.x == 0 && blockIdx.x == 0 &&
       threadIdx.y == 0 && blockIdx.y == 0 &&
       threadIdx.z == 0 && blockIdx.z == 0){
           printf("%d %d %d %d %d %d\n", gridDim.x, gridDim.y, gridDim.z, 
                                         blockDim.x, blockDim.y, blockDim.z);
       }
}

int main(){
    dim3 grid(2, 3, 4);
    dim3 block(5, 6, 7);
    dkernal <<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}
