#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 8000  

__global__ void fun(int *arr){
    unsigned id = threadIdx.x;
    if(id < N) arr[id] = 0;
}

__global__ void check(int* arr){
    unsigned id = threadIdx.x;
    if(id < N) arr[id] += id;
} 

int main(){
    int *da;
    int i;
    int a[N];
    hipMalloc(&da, N * sizeof(int));
    fun<<<1, N>>>(da);
    check<<<1, N>>>(da); 
    hipMemcpy(a, da, N * sizeof(int), hipMemcpyDeviceToHost);
    for(i = 0 ; i < N ; ++i)
    {
        printf("%d ", a[i]);
    }
    return 0;
}
